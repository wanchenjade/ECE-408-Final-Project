#include <iostream>
#include <algorithm>
#include <vector>
#include <iomanip>
#include <hip/hip_runtime.h>

#include <sys/time.h>

using std::swap;
using std::cout;
using std::vector;

inline double seconds() {
  struct timeval tp;
  struct timezone tzp;
  int i = gettimeofday(&tp, &tzp);
  return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

#define BLOCK_SIZE 128
#define CHECK(call)                                                            \
  {                                                                            \
    const hipError_t error = call;                                            \
    if (error != hipSuccess) {                                                \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                   \
      fprintf(stderr, "code: %d, reason: %s\n", error,                         \
              hipGetErrorString(error));                                      \
      exit(1);                                                                 \
    }                                                                          \
  }

struct Node {
  int id;
  int num_nodes;          // number of nodes in the subtree
  Node *left;             // left subtree
  Node *right;            // right subtree
  float total_length;     // total length of the subtree
  float branch_length[2]; // lengths of left and right subtrees
  Node(int _id, int _num_nodes, float _length, Node *_left, Node *_right,
       float length1, float length2)
      : id(_id), num_nodes(_num_nodes), left(_left), right(_right),
        total_length(_length) {
    branch_length[0] = length1;
    branch_length[1] = length2;
  }
};

template <unsigned int blockSize>
__global__ void getMin(float *input, int *input_idx, int n, float *output_val,
                       int *output_idx) {
  __shared__ float smem_val[blockSize];
  __shared__ int smem_idx[blockSize];

  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int i = tx + bx * blockSize * 8;

  float min_val = INFINITY;
  int min_idx = i;

  if (i < n) {
    float a1, a2, a3, a4, a5, a6, a7, a8;
    a1 = input[i];
    min_val = a1;
    min_idx = i;

    i += blockSize;
    a2 = i < n ? input[i] : INFINITY;
    if (a2 < min_val) {
      min_val = a2;
      min_idx = i;
    }

    i += blockSize;
    a3 = i < n ? input[i] : INFINITY;
    if (a3 < min_val) {
      min_val = a3;
      min_idx = i;
    }

    i += blockSize;
    a4 = i < n ? input[i] : INFINITY;
    if (a4 < min_val) {
      min_val = a4;
      min_idx = i;
    }

    i += blockSize;
    a5 = i < n ? input[i] : INFINITY;
    if (a5 < min_val) {
      min_val = a5;
      min_idx = i;
    }

    i += blockSize;
    a6 = i < n ? input[i] : INFINITY;
    if (a6 < min_val) {
      min_val = a6;
      min_idx = i;
    }

    i += blockSize;
    a7 = i < n ? input[i] : INFINITY;
    if (a7 < min_val) {
      min_val = a7;
      min_idx = i;
    }

    i += blockSize;
    a8 = i < n ? input[i] : INFINITY;
    if (a8 < min_val) {
      min_val = a8;
      min_idx = i;
    }
  }

  smem_val[tx] = min_val;
  smem_idx[tx] = min_idx;
  __syncthreads();

  // in-place reduction in shared memory
  if (blockSize >= 1024 && tx < 512 && smem_val[tx + 512] < min_val) {
    smem_val[tx] = min_val = smem_val[tx + 512];
    smem_idx[tx] = min_idx = smem_idx[tx + 512];
  }
  __syncthreads();

  if (blockSize >= 512 && tx < 256 && smem_val[tx + 256] < min_val) {
    smem_val[tx] = min_val = smem_val[tx + 256];
    smem_idx[tx] = min_idx = smem_idx[tx + 256];
  }
  __syncthreads();

  if (blockSize >= 256 && tx < 128 && smem_val[tx + 128] < min_val) {
    smem_val[tx] = min_val = smem_val[tx + 128];
    smem_idx[tx] = min_idx = smem_idx[tx + 128];
  }
  __syncthreads();

  if (blockSize >= 128 && tx < 64 && smem_val[tx + 64] < min_val) {
    smem_val[tx] = min_val = smem_val[tx + 64];
    smem_idx[tx] = min_idx = smem_idx[tx + 64];
  }
  __syncthreads();

  // unrolling warp
  if (tx < 32) {
    volatile float *vsmem_val = smem_val;
    volatile int *vsmem_idx = smem_idx;
    if (blockSize >= 64 && vsmem_val[tx + 32] < min_val) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 32];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 32];
    }
    if (blockSize >= 32 && vsmem_val[tx + 16] < min_val) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 16];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 16];
    }
    if (blockSize >= 16 && vsmem_val[tx + 8] < min_val) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 8];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 8];
    }
    if (blockSize >= 8 && vsmem_val[tx + 4] < min_val) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 4];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 4];
    }
    if (blockSize >= 4 && vsmem_val[tx + 2] < min_val) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 2];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 2];
    }
    if (blockSize >= 2 && vsmem_val[tx + 1] < min_val) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 1];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 1];
    }
  }

  if (tx == 0) {
    output_val[bx] = min_val;
    output_idx[bx] = (input_idx == nullptr) ? min_idx : input_idx[min_idx];
  }
}

__global__ void update(float *mat, int n, int idx1, int idx2, int num_nodes1,
                       int num_nodes2) {
  int tx = threadIdx.x;
  int i = tx + blockDim.x * blockIdx.x;
  if (i >= n) {
    return;
  }
  if (i == idx2) {
    mat[n * idx1 + i] = INFINITY;
    mat[n * i + idx1] = INFINITY;
    return;
  }
  float val = mat[n * idx1 + i];
  if (isinf(val)) {
    return;
  }
  int total_nodes = num_nodes1 + num_nodes2;
  float new_val =
      (val * num_nodes1 + mat[n * idx2 + i] * num_nodes2) / total_nodes;
  mat[n * idx1 + i] = new_val;
  mat[n * idx2 + i] = INFINITY;
  mat[n * i + idx1] = new_val;
  mat[n * i + idx2] = INFINITY;
}

class UPGMA {
public:
  UPGMA(float *_mat, int _num_seqs) {
    h_mat = _mat;
    num_seqs = _num_seqs;

    int n = num_seqs * num_seqs;
    int n_out_level0 = ceil((float)n / (BLOCK_SIZE * 8));
    int n_out_level1 = ceil((float)n_out_level0 / (BLOCK_SIZE * 8));

    // Allocate host variables
    // Result values after level 1 reduction for final reduction
    float *h_val_level1 = (float *)malloc(sizeof(float) * n_out_level1);
    // Result indexes after level 1 reduction for final reduction
    int *h_idx_level1 = (int *)malloc(sizeof(int) * n_out_level1);

    // Allocate device variables
    float *d_mat;                       // Device matrix
    float *d_val_level0, *d_val_level1; // Device result values
    int *d_idx_level0, *d_idx_level1;   // Device index values
    CHECK(hipMalloc((void **)&d_mat, sizeof(float) * n));
    CHECK(hipMalloc((void **)&d_val_level0, sizeof(float) * n_out_level0));
    CHECK(hipMalloc((void **)&d_idx_level0, sizeof(int) * n_out_level0));
    CHECK(hipMalloc((void **)&d_val_level1, sizeof(float) * n_out_level1));
    CHECK(hipMalloc((void **)&d_idx_level1, sizeof(int) * n_out_level1));

    // Copy from host to device
    CHECK(hipMemcpy(d_mat, h_mat, sizeof(float) * n, hipMemcpyHostToDevice));

    vector<Node *> nodes(num_seqs);
    for (int i = 0; i < num_seqs; ++i) {
      nodes[i] = new Node(i, 1, 0.0f, nullptr, nullptr, 0.0f, 0.0f);
    }

    for (int remain = num_seqs; remain >= 2; --remain) {
      // Reduction round 1
      getMin<BLOCK_SIZE><<<n_out_level0, BLOCK_SIZE>>>(
          d_mat, nullptr, n, d_val_level0, d_idx_level0);

      CHECK(hipDeviceSynchronize());

      // Reduction round 2
      getMin<BLOCK_SIZE><<<n_out_level1, BLOCK_SIZE>>>(
          d_val_level0, d_idx_level0, n_out_level0, d_val_level1, d_idx_level1);

      CHECK(hipDeviceSynchronize());

      // Copy results and indexes back
      CHECK(hipMemcpy(h_val_level1, d_val_level1, sizeof(float) * n_out_level1,
                       hipMemcpyDeviceToHost));
      CHECK(hipMemcpy(h_idx_level1, d_idx_level1, sizeof(int) * n_out_level1,
                       hipMemcpyDeviceToHost));

      float val = h_val_level1[0];
      int idx = h_idx_level1[0];
      for (int i = 0; i < n_out_level1; ++i) {
        if (h_val_level1[i] < val) {
          val = h_val_level1[i];
          idx = h_idx_level1[i];
        }
      }

      int idx1 = idx / num_seqs;
      int idx2 = idx % num_seqs;
      if (idx1 > idx2) {
        swap(idx1, idx2);
      }

      // Update
      update<<<ceil(num_seqs / (float)BLOCK_SIZE), BLOCK_SIZE>>>(
          d_mat, num_seqs, idx1, idx2, nodes[idx1]->num_nodes,
          nodes[idx2]->num_nodes);

      // Combine two nodes into one
      float length = val;
      root = new Node(-1, nodes[idx1]->num_nodes + nodes[idx2]->num_nodes,
                      length / 2, nodes[idx1], nodes[idx2],
                      length / 2 - nodes[idx1]->total_length,
                      length / 2 - nodes[idx2]->total_length);

      nodes[idx1] = root;
      nodes[idx2] = nullptr;

      CHECK(hipDeviceSynchronize());
    }

    // Free device memory
    CHECK(hipFree(d_mat));
    CHECK(hipFree(d_val_level0));
    CHECK(hipFree(d_idx_level0));
    CHECK(hipFree(d_val_level1));
    CHECK(hipFree(d_idx_level1));

    // Free host memory
    free(h_val_level1);
    free(h_idx_level1);
  }

  ~UPGMA() { cleanup(root); }

  void print() {
    print(root);
    cout << "\n";
  }

private:
  float *h_mat;
  int num_seqs;
  Node *root;

  void cleanup(Node *node) {
    if (node == nullptr) {
      return;
    }
    cleanup(node->left);
    cleanup(node->right);
    delete node;
  }

  void print(Node *node) {
    // Reach the leaf
    if (node->left == nullptr && node->right == nullptr) {
      cout << "A" + std::to_string(node->id);
      return;
    }
    cout << "(";
    print(node->left);
    cout << ": " << std::fixed << node->branch_length[0] << ", ";
    print(node->right);
    cout << ": " << std::fixed << node->branch_length[1] << ")";
  }
};

int main(int argc, char *argv[]) {
#if 0
  // This is the test case
  // The tree should have the same shape as the tree on this page (Source tab)
  // http://www.southampton.ac.uk/~re1u06/teaching/upgma/
  // Convention: A0 = A, A1 = B, A2 = C, A3 = D, A4 = E, A5 = F, A6 = G
  const int num_seqs = 7;
  float a[num_seqs][num_seqs]{
      {INFINITY, 19.0f, 27.0f, 8.0f, 33.0f, 18.0f, 13.0f},
      {19.0f, INFINITY, 31.0f, 18.0f, 36.0f, 1.0f, 13.0f},
      {27.0f, 31.0f, INFINITY, 26.0f, 41.0f, 32.0f, 29.0f},
      {8.0f, 18.0f, 26.0f, INFINITY, 31.0f, 17.0f, 14.0f},
      {33.0f, 36.0f, 41.0f, 31.0f, INFINITY, 35.0f, 28.0f},
      {18.0f, 1.0f, 32.0f, 17.0f, 35.0f, INFINITY, 12.0f},
      {13.0f, 13.0f, 29.0f, 14.0f, 28.0f, 12.0f, INFINITY}};
  UPGMA upgma((float *)a, num_seqs);
  upgma.print();
#else
  // This is mock data to test with large matrix
  if (argc != 2) {
    cout << "Usage: " << argv[0] << " number\n";
    exit(-1);
  }
  const int num_seqs = atoi(argv[1]);
  float *a = new float[num_seqs * num_seqs];
  srand(0);
  for (int i = 0; i < num_seqs; ++i) {
    for (int j = 0; j < i; ++j) {
      a[i * num_seqs + j] = rand() / (float)RAND_MAX;
      a[j * num_seqs + i] = a[i * num_seqs + j];
    }
    a[i * num_seqs + i] = INFINITY;
  }

  double start = seconds();
  UPGMA upgma(a, num_seqs);
  double elapsed = seconds() - start;
  upgma.print();
  //cout << "Time to reconstruct the tree: " << elapsed << "\n";
  delete a;
#endif
  return 0;
}
